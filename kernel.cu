
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
        int i = threadIdx.x + blockDim.x * blockIdx.x;

        int dimensions = dim*dim;

        if(i < dimensions)
        {
            C[i] = A[i] + B[i];
        }


    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE

    int number_of_blocks = ((dim*dim) - 1)/BLOCK_SIZE + 1;

    dim3 Dimension_Grid(number_of_blocks, 1, 1);
    dim3 Dimension_Block(BLOCK_SIZE,  1, 1);
    /*************************************************************************/
	
	// Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	matAdd<<<Dimension_Grid,Dimension_Block>>>(dim,A,B,C);
    /*************************************************************************/

}

